#include <stdio.h>
#include <hip/hip_runtime.h>
#include<sys/time.h>
#include <time.h>
#include<math.h>


// Kernel that executes on the CUDA device
__global__ void square_array(float *a, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx<N) a[idx] = sqrtf(powf(2,a[idx]) + powf(a[idx],3));
}


// main routine that executes on the host
int main(void)
{
struct timeval iniciototal, finaltotal;
    gettimeofday(&iniciototal, NULL);
    int N, exec;  // Number of elements in arrays
    struct timeval inicio, final;
    int tmili,tmilifim;
    float media, soma ;
    int tam;
    for (N = 10; N <= 18; N++)
    {
    soma=0;
    tam = pow(2,N);
        for(exec =1; exec <= 10; exec++)
        {
            clock_t start, end;
            double cpu_time_used;


            start = clock();

            gettimeofday(&inicio, NULL);


            float *a_h, *a_d;  // Pointer to host & device arrays
            size_t size = tam * sizeof(float);
            a_h = (float *)malloc(size);        // Allocate array on host


            hipMalloc((void **) &a_d, size);   // Allocate array on device
            // Initialize host array and copy it to CUDA device
            for (int i=0; i<tam; i++) a_h[i] = (float)i;
            hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
            // Do calculation on device:
            int block_size = 32;
            int n_blocks = tam/block_size + (tam%block_size == 0 ? 0:1);


            square_array <<< n_blocks, block_size >>> (a_d, tam);

            // Retrieve result from device and store it in host array
            hipMemcpy(a_h, a_d, sizeof(float)*tam, hipMemcpyDeviceToHost);






            //printf("%d\n", n_blocks);
            // Print results
            //for (int i=0; i<N; i++) printf("%d %f\n", i, a_h[i]);
            // Cleanup
            free(a_h);
            hipFree(a_d);

            end = clock();
            cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
            printf("\nTempo Clock: %f secs", cpu_time_used);
            gettimeofday(&final, NULL);
            tmili = (int) (1000 * (final.tv_sec - inicio.tv_sec) + (final.tv_usec - inicio.tv_usec) / 1000);

            soma+=tmili;

        }

        media = soma/10;
        printf("tamanho: %d \t tempo decorrido: %f\n", tam, media);
    }

    gettimeofday(&finaltotal, NULL);
            tmilifim = (int) (1000 * (finaltotal.tv_sec - iniciototal.tv_sec) + (finaltotal.tv_usec - iniciototal.tv_usec) / 1000);
            printf("tempo total decorrido: %d\n", tmilifim);
}
